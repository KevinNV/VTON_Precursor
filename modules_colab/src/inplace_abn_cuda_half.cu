#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_fp16.h>
#include <vector>
#include "utils/checks.h"
#include "utils/cuda.cuh"
#include "inplace_abn.h"
#include <ATen/cuda/HIPContext.h>

// ------------------- DEVICE STRUCTS ------------------- //

struct SumOpH {
  __device__ SumOpH(const half *t, int c, int s) : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    return __half2float(tensor[(batch * chn + plane) * sp + n]);
  }
  const half *tensor;
  const int chn, sp;
};

struct VarOpH {
  __device__ VarOpH(float m, const half *t, int c, int s) : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    float t = __half2float(tensor[(batch * chn + plane) * sp + n]);
    return (t - mean) * (t - mean);
  }
  const float mean;
  const half *tensor;
  const int chn, sp;
};

struct GradOpH {
  __device__ GradOpH(float weight, float bias, const half *z, const half *dz, int c, int s)
      : weight(weight), bias(bias), z(z), dz(dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<float> operator()(int batch, int plane, int n) {
    float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - bias) / weight;
    float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
    return Pair<float>(_dz, _y * _dz);
  }
  const float weight, bias;
  const half *z, *dz;
  const int chn, sp;
};

// ------------------- MEAN VAR ------------------- //

__global__ void mean_var_kernel_h(const half *x, float *mean, float *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  float norm = 1.f / static_cast<float>(num * sp);
  float _mean = reduce<float, SumOpH>(SumOpH(x, chn, sp), plane, num, sp) * norm;
  __syncthreads();
  float _var = reduce<float, VarOpH>(VarOpH(_mean, x, chn, sp), plane, num, sp) * norm;
  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda_h(at::Tensor x) {
  CHECK_CUDA_INPUT(x);
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);
  auto mean = at::empty({chn}, x.options().dtype(at::kFloat));
  auto var = at::empty({chn}, x.options().dtype(at::kFloat));
  dim3 blocks(chn), threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  mean_var_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<const half*>(x.data_ptr<at::Half>()),
      mean.data_ptr<float>(),
      var.data_ptr<float>(),
      num, chn, sp);
  return {mean, var};
}

// ------------------- FORWARD ------------------- //

__global__ void forward_kernel_h(half *x, const float *mean, const float *var, const float *weight, const float *bias,
                                 bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;
  const float _mean = mean[plane];
  const float _var = var[plane];
  const float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  const float _bias = affine ? bias[plane] : 0.f;
  const float mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      half *x_ptr = x + (batch * chn + plane) * sp + n;
      float _x = __half2float(*x_ptr);
      float _y = (_x - _mean) * mul + _bias;
      *x_ptr = __float2half(_y);
    }
  }
}

at::Tensor forward_cuda_h(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                          bool affine, float eps) {
  CHECK_CUDA_INPUT(x); CHECK_CUDA_INPUT(mean); CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight); CHECK_CUDA_INPUT(bias);
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);
  dim3 blocks(chn), threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  forward_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<half*>(x.data_ptr<at::Half>()),
      mean.data_ptr<float>(), var.data_ptr<float>(),
      weight.data_ptr<float>(), bias.data_ptr<float>(),
      affine, eps, num, chn, sp);
  return x;
}

// ------------------- LEAKY RELU BACKWARD ------------------- //

__global__ void leaky_relu_backward_impl_h(half *z, half *dz, float slope, int64_t count) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
    float _z = __half2float(z[i]);
    if (_z < 0.0f) {
      dz[i] = __float2half(__half2float(dz[i]) * slope);
      z[i] = __float2half(_z / slope);
    }
  }
}

void leaky_relu_backward_cuda_h(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);

  int64_t count = z.numel();
  dim3 threads(getNumThreads(count));
  dim3 blocks((count + threads.x - 1) / threads.x);
  auto stream = at::cuda::getCurrentCUDAStream();

  leaky_relu_backward_impl_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<half*>(z.data_ptr<at::Half>()),
      reinterpret_cast<half*>(dz.data_ptr<at::Half>()),
      slope, count);
}

__global__ void edz_eydz_kernel_h(const half *z, const half *dz, const float *weight, const float *bias,
                                  float *edz, float *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;

  Pair<float> res = reduce<Pair<float>, GradOpH>(GradOpH(_weight, _bias, z, dz, chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                        bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty({chn}, z.options().dtype(at::kFloat));
  auto eydz = at::empty({chn}, z.options().dtype(at::kFloat));

  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();

  edz_eydz_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<const half*>(z.data_ptr<at::Half>()),
      reinterpret_cast<const half*>(dz.data_ptr<at::Half>()),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      edz.data_ptr<float>(),
      eydz.data_ptr<float>(),
      affine, eps, num, chn, sp
  );

  return {edz, eydz};
}

__global__ void backward_kernel_h(const half *z, const half *dz, const float *var,
                                  const float *weight, const float *bias,
                                  const float *edz, const float *eydz,
                                  half *dx, bool affine, float eps,
                                  int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;
  float _var = var[plane];
  float _edz = edz[plane];
  float _eydz = eydz[plane];

  float _mul = _weight * rsqrt(_var + eps);
  float count = float(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      int idx = (batch * chn + plane) * sp + n;

      float _dz = __half2float(dz[idx]);
      float _y = (__half2float(z[idx]) - _bias) / _weight;

      float _dx = (_dz - _edz / count - _y * _eydz / count) * _mul;
      dx[idx] = __float2half(_dx);
    }
  }
}

at::Tensor backward_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                           at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);
  CHECK_CUDA_INPUT(edz);
  CHECK_CUDA_INPUT(eydz);

  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::empty_like(z);

  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();

  backward_kernel_h<<<blocks, threads, 0, stream>>>(
      reinterpret_cast<const half*>(z.data_ptr<at::Half>()),
      reinterpret_cast<const half*>(dz.data_ptr<at::Half>()),
      var.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      edz.data_ptr<float>(),
      eydz.data_ptr<float>(),
      reinterpret_cast<half*>(dx.data_ptr<at::Half>()),
      affine, eps, num, chn, sp
  );

  return dx;
}


